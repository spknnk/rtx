#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <csignal>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <iterator>
#include <sstream>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"
#include "mpi.h"
#include "omp.h"

#define NBLOCKS dim3(128, 128)
#define NTHREADS dim3(16, 16)

#define SSSAx 2
#define PHONG_INTENS 5
#define PHONG_KD 0.8
#define PHONG_KS 0.8

#define FATAL(description)                                      \
    do {                                                        \
        std::cout << "Error in " << __FILE__ << ":" << __LINE__ \
                  << ". Message: " << description << std::endl; \
        MPI_Finalize();                                         \
        exit(0);                                                \
    } while (0)

#define CSC(call)                  \
    do {                                    \
        hipError_t res = call;             \
        if (res != hipSuccess) {           \
            FATAL(hipGetErrorString(res)); \
        }                                   \
    } while (0)

#define CHECK_MPI(call)                        \
    do {                                       \
        int res = call;                        \
        if (res != MPI_SUCCESS) {              \
            char desc[MPI_MAX_ERROR_STRING];   \
            int len;                           \
            MPI_Error_string(res, desc, &len); \
            FATAL(desc);                       \
        }                                      \
    } while (0)

#define OpenMP 0
#define CUDA 1
int parallelizationMode;

void parse_flags(int argc, char *argv[]) {
   
    if (argc < 2) return;
    if (argc == 2) {
        if (strcmp(argv[1], "--gpu") == 0) {
            parallelizationMode = CUDA;
        }
        
        if (strcmp(argv[1], "--cpu") == 0) {
            parallelizationMode = OpenMP;
        }
        
        else if ((strcmp(argv[1], "--cpu") != 0) and (strcmp(argv[1], "--gpu") != 0)) {
            std::cout << "Unknown args" << std::endl;
            MPI_Finalize();
            exit(0);
        }
    }
    if (argc > 2) {
        std::cout << "A lot of args" << std::endl;
        MPI_Finalize();
        exit(0);
    }
}

struct MPIContext {
    MPIContext(int *argc, char ***argv) { CHECK_MPI(MPI_Init(argc, argv)); }
    ~MPIContext() {
        CHECK_MPI(MPI_Barrier(MPI_COMM_WORLD));
        CHECK_MPI(MPI_Finalize());
    }
};

template <typename T>
struct Vector3 {
    T x, y, z;

    __host__ __device__ Vector3(T x = T{}, T y = T{}, T z = T{})
        : x(x), y(y), z(z) {}

    friend std::istream &operator>>(std::istream &is, Vector3 &v) {
        is >> v.x >> v.y >> v.z;
        return is;
    }
};

struct CameraMovement {
    double r0, z0, phi0, ar, az, wr, wz, wphi, pr, pz;

    friend std::istream &operator>>(std::istream &is, CameraMovement &p) {
        is >> p.r0 >> p.z0 >> p.phi0 >> p.ar >> p.az >> p.wr >> p.wz >> p.wphi >> p.pr >> p.pz;
        return is;
    }
};

struct FigureParams {
    Vector3<double> center, color;
    double radius, k_refl, k_refr;
    int lights_num;

    friend std::istream &operator>>(std::istream &is, FigureParams &p) {
        is >> p.center >> p.color >> p.radius >> p.k_refl >> p.k_refr >> p.lights_num;
        return is;
    }
};

struct FloorParams {
    Vector3<double> a, b, c, d, color;
    double k_refl;
    std::string texture_path;

    friend std::istream &operator>>(std::istream &is, FloorParams &p) {
        is >> p.a >> p.b >> p.c >> p.d >> p.texture_path >> p.color >> p.k_refl;
        return is;
    }
};

struct LightParams {
    Vector3<double> pos;
    Vector3<double> color;

    friend std::istream &operator>>(std::istream &is, LightParams &p) {
        is >> p.pos >> p.color;
        return is;
    }
};

struct Params {
    int nframes, w, h, lights_num;
    double angle;
    CameraMovement camera_center, camera_dir;
    FigureParams hex, octa, icos;
    FloorParams floor;
    
    std::string output_pattern;
    std::vector<LightParams> lights;

    friend std::istream &operator>>(std::istream &is, Params &p) {
        is >> p.nframes >> p.output_pattern >> p.w >> p.h >> p.angle >> p.camera_center >> p.camera_dir >> p.hex >> p.octa >> p.icos >> p.floor >> p.lights_num;
        p.lights.resize(p.lights_num);
        for (auto &it : p.lights) is >> it;
        return is;
    }
};

struct Triangle {
    Vector3<double> a, b, c, color;
};

template <typename T>
__host__ __device__ T min(const T &a, const T &b) {
    if (a < b) return a;
    return b;
}

template <typename T>
__host__ __device__ T max(const T &a, const T &b) {
    if (a > b) return a;
    return b;
}

__host__ __device__ double dot_product(const Vector3<double> &a, const Vector3<double> &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ Vector3<double> cross_product(const Vector3<double> &a, const Vector3<double> &b) {
    return {a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,a.x * b.y - a.y * b.x};
}

__host__ __device__ double norm(const Vector3<double> &v) {
    return sqrt(dot_product(v, v));
}

__host__ __device__ Vector3<double> normalize(const Vector3<double> &v) {
    double l = norm(v);
    return {v.x / l, v.y / l, v.z / l};
}

__host__ __device__ Vector3<double> diff(const Vector3<double> &a, const Vector3<double> &b) {
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__host__ __device__ Vector3<double> add(const Vector3<double> &a, const Vector3<double> &b) {
    return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__host__ __device__ Vector3<double> mult(const Vector3<double> &a, const Vector3<double> &b) {
    return {b.x * a.x, b.y * a.y, b.z * a.z};
}

__host__ __device__ Vector3<double> mult(const Vector3<double> &a, double k) {
    return {k * a.x, k * a.y, k * a.z};
}

__host__ __device__ Vector3<double> mult(const Vector3<double> &a, const Vector3<double> &b, const Vector3<double> &c, const Vector3<double> &d) {
    return {a.x * d.x + b.x * d.y + c.x * d.z,
            a.y * d.x + b.y * d.y + c.y * d.z,
            a.z * d.x + b.z * d.y + c.z * d.z};
}

__host__ __device__ Vector3<double> inverse(const Vector3<double> &v) {
    return {-v.x, -v.y, -v.z};
}

__host__ __device__ Vector3<double> div(const Vector3<double> &a, double k) {
    return {a.x / k, a.y / k, a.z / k};
}

__host__ __device__ Vector3<double> reflect(const Vector3<double> &v, const Vector3<double> &n) {
    return diff(v, mult(n, 2.0 * dot_product(v, n)));
}

__host__ __device__ uchar4 color_from_normalized(const Vector3<double> &v) {
    double x = min(v.x, 1.);
    x = max(x, 0.);
    double y = min(v.y, 1.);
    y = max(y, 0.);
    double z = min(v.z, 1.);
    z = max(z, 0.);
    return make_uchar4(255. * x, 255. * y, 255. * z, 0u);
}

std::vector<std::string> split_string(const std::string &s, char d) {
    std::vector<std::string> result;
    std::stringstream ss(s);
    std::string word;
    while (getline(ss, word, d)) {
        result.push_back(word);
    }
    return result;
}

void importObj(std::vector<Triangle> &scene_Triangles, const std::string &filepath, const FigureParams &fp) {
    std::ifstream is(filepath);
    if (!is) {
        std::string desc = "can't open " + filepath;
        FATAL(desc);
    }

    /*
    std::vector<Triangle> triangle_list(10);
    triangle_list[0] = Triangle{Vector3<double> {}, Vector3<double> {}, Vector3<double> {}, fp.color}
    */

    double r = 0;
    std::vector<Vector3<double>> vertices;
    std::vector<Triangle> figure_Triangles;
    std::string line;
    while (std::getline(is, line)) {
        std::vector<std::string> buffer = split_string(line, ' ');
        if (line.empty()) {
            continue;
        } else if (buffer[0] == "v") {
            double x = std::stod(buffer[2]);
            double y = std::stod(buffer[3]);
            double z = std::stod(buffer[4]);

            vertices.push_back({x, y, z});
        } else if (buffer[0] == "f") {
            std::vector<std::string> indexes = split_string(buffer[1], '/');
            Vector3<double> a = vertices[std::stoi(indexes[0]) - 1];
            indexes = split_string(buffer[2], '/');
            Vector3<double> b = vertices[std::stoi(indexes[0]) - 1];
            indexes = split_string(buffer[3], '/');
            Vector3<double> c = vertices[std::stoi(indexes[0]) - 1];

            r = max(r, norm(a));
            r = max(r, norm(b));
            r = max(r, norm(c));

            figure_Triangles.push_back(Triangle{a, b, c, fp.color});
        }
    }

    double radius = fp.radius / r;
    for (auto &single_triangle : figure_Triangles) {
        scene_Triangles.push_back({add(mult(single_triangle.a, radius), fp.center), add(mult(single_triangle.b, radius), fp.center), add(mult(single_triangle.c, radius), fp.center), single_triangle.color});
    }
}



void add_floor_to_scene(std::vector<Triangle> &scene_Triangles, const FloorParams &fp) {
    scene_Triangles.push_back({fp.c, fp.b, fp.a, fp.color});
    scene_Triangles.push_back({fp.a, fp.d, fp.c, fp.color});
}

struct Mat3d {
    double m[3][3];
    __host__ __device__ Mat3d(double m11 = 0, double m12 = 0, double m13 = 0,
                              double m21 = 0, double m22 = 0, double m23 = 0,
                              double m31 = 0, double m32 = 0, double m33 = 0) {
        m[0][0] = m11;
        m[0][1] = m12;
        m[0][2] = m13;
        m[1][0] = m21;
        m[1][1] = m22;
        m[1][2] = m23;
        m[2][0] = m31;
        m[2][1] = m32;
        m[2][2] = m33;
    }
};

__host__ __device__ double det(const Mat3d &m) {
    return m.m[0][0] * m.m[1][1] * m.m[2][2] +
           m.m[1][0] * m.m[0][2] * m.m[2][1] +
           m.m[2][0] * m.m[0][1] * m.m[1][2] -
           m.m[0][2] * m.m[1][1] * m.m[2][0] -
           m.m[0][0] * m.m[1][2] * m.m[2][1] -
           m.m[0][1] * m.m[1][0] * m.m[2][2];
}

__host__ __device__ Mat3d inverse(const Mat3d &m) {
    double d = det(m);

    double m11 = (m.m[1][1] * m.m[2][2] - m.m[2][1] * m.m[1][2]) / d;
    double m12 = (m.m[2][1] * m.m[0][2] - m.m[0][1] * m.m[2][2]) / d;
    double m13 = (m.m[0][1] * m.m[1][2] - m.m[1][1] * m.m[0][2]) / d;

    double m21 = (m.m[2][0] * m.m[1][2] - m.m[1][0] * m.m[2][2]) / d;
    double m22 = (m.m[0][0] * m.m[2][2] - m.m[2][0] * m.m[0][2]) / d;
    double m23 = (m.m[1][0] * m.m[0][2] - m.m[0][0] * m.m[1][2]) / d;

    double m31 = (m.m[1][0] * m.m[2][1] - m.m[2][0] * m.m[1][1]) / d;
    double m32 = (m.m[2][0] * m.m[0][1] - m.m[0][0] * m.m[2][1]) / d;
    double m33 = (m.m[0][0] * m.m[1][1] - m.m[1][0] * m.m[0][1]) / d;

    return Mat3d(m11, m12, m13, m21, m22, m23, m31, m32, m33);
}

__host__ __device__ Vector3<double> mult(const Mat3d &m, const Vector3<double> &v) {
    Vector3<double> res;
    res.x = m.m[0][0] * v.x + m.m[0][1] * v.y + m.m[0][2] * v.z;
    res.y = m.m[1][0] * v.x + m.m[1][1] * v.y + m.m[1][2] * v.z;
    res.z = m.m[2][0] * v.x + m.m[2][1] * v.y + m.m[2][2] * v.z;
    return res;
}

__host__ __device__ void triangle_intersection(const Vector3<double> &origin, const Vector3<double> &dir, const Triangle &Triangle, double *t, double *u, double *v) {
    Vector3<double> e1 = diff(Triangle.b, Triangle.a);
    Vector3<double> e2 = diff(Triangle.c, Triangle.a);

    Mat3d m(-dir.x, e1.x, e2.x, -dir.y, e1.y, e2.y, -dir.z, e1.z, e2.z);
    Vector3<double> tmp = mult(inverse(m), diff(origin, Triangle.a));

    *t = tmp.x;
    *u = tmp.y;
    *v = tmp.z;
}

__host__ __device__ bool shadow_ray_hit(const Vector3<double> &origin, const Vector3<double> &dir, const Triangle *scene_Triangles, int nTriangles, double *hit_t) {
    double t_min = 1 / 0.;
    bool hit = false;
    for (int i = 0; i < nTriangles; ++i) {
        auto Triangle = scene_Triangles[i];
        double t, u, v;
        triangle_intersection(origin, dir, Triangle, &t, &u, &v);
        if (u >= 0.0 && v >= 0.0 && u + v <= 1.0 && t > 0.0) {
            if (t < t_min) {
                t_min = t;
            }
            hit = true;
        }
    }
    *hit_t = t_min;
    return hit;
}

__host__ __device__ Vector3<double> phong_model(const Vector3<double> &pos, const Vector3<double> &dir, const Triangle &TriangleObj, const Triangle *scene_Triangles, int nTriangles, const LightParams *lights, int lights_num) {
    Vector3<double> normal = normalize(cross_product(diff(TriangleObj.b, TriangleObj.a), diff(TriangleObj.c, TriangleObj.a)));

    Vector3<double> ambient{0.1, 0.1, 0.1};
    Vector3<double> diffuse{0., 0., 0.};
    Vector3<double> specular{0., 0., 0.};

    for (int i = 0; i < lights_num; ++i) {
        Vector3<double> light_pos = lights[i].pos;
        Vector3<double> L = diff(light_pos, pos);
        double d = norm(L);
        L = normalize(L);

        double hit_t = 0.0;
        if (shadow_ray_hit(light_pos, inverse(L), scene_Triangles, nTriangles, &hit_t) && (hit_t > d || (hit_t > d || (d - hit_t < 0.0005)))) {
            double k = PHONG_INTENS / (d + 0.001f);
            diffuse = add(diffuse, mult(lights[i].color, max(PHONG_KD * k * dot_product(L, normal), 0.0)));

            Vector3<double> R = normalize(reflect(inverse(L), normal));
            Vector3<double> S = inverse(dir);
            specular = add(specular, mult(lights[i].color, PHONG_KS * k * std::pow(max(dot_product(R, S), 0.0), 32)));
        }
    }
    return add(add(mult(ambient, TriangleObj.color), mult(diffuse, TriangleObj.color)), mult(specular, TriangleObj.color));
}

__host__ __device__ uchar4 ray(const Vector3<double> &pos, const Vector3<double> &dir, const Triangle *scene_Triangles, int nTriangles, LightParams *lights, int lights_num) {
    int k, k_min = -1;
    double ts_min;
    for (k = 0; k < nTriangles; k++) {
        Vector3<double> e1 = diff(scene_Triangles[k].b, scene_Triangles[k].a);
        Vector3<double> e2 = diff(scene_Triangles[k].c, scene_Triangles[k].a);
        Vector3<double> p = cross_product(dir, e2);
        double div = dot_product(p, e1);
        if (fabs(div) < 1e-10) continue;
        Vector3<double> t = diff(pos, scene_Triangles[k].a);
        double u = dot_product(p, t) / div;
        if (u < 0.0 || u > 1.0) continue;
        Vector3<double> q = cross_product(t, e1);
        double v = dot_product(q, dir) / div;
        if (v < 0.0 || v + u > 1.0) continue;
        double ts = dot_product(q, e2) / div;
        if (ts < 0.0) continue;
        if (k_min == -1 || ts < ts_min) {
            k_min = k;
            ts_min = ts;
        }
    }
    if (k_min == -1) return {0, 0, 0, 0};
    return color_from_normalized(phong_model(add(mult(dir, ts_min), pos), dir, scene_Triangles[k_min], scene_Triangles, nTriangles, lights, lights_num));
}

void getRenderOMP(uchar4 *data, int w, int h, Vector3<double> pc, Vector3<double> pv, double angle, const Triangle *scene_Triangles, int nTriangles, LightParams *lights, int lights_num) {
    double dw = 2.0 / (w - 1.0);
    double dh = 2.0 / (h - 1.0);
    double z = 1.0 / tan(angle * M_PI / 360.0);
    Vector3<double> bz = normalize(diff(pv, pc));
    Vector3<double> bx = normalize(cross_product(bz, {0.0, 0.0, 1.0}));
    Vector3<double> by = normalize(cross_product(bx, bz));
#pragma omp parallel for
    for (int pix = 0; pix < w * h; ++pix) {
        int i = pix % w;
        int j = pix / w;
        Vector3<double> v = {-1.0 + dw * i, (-1.0 + dh * j) * h / w, z};
        Vector3<double> dir = mult(bx, by, bz, v);
        data[(h - 1 - j) * w + i] = ray(pc, normalize(dir), scene_Triangles, nTriangles, lights, lights_num);
    }
}

__global__ void getRenderCUDA(uchar4 *data, int w, int h, Vector3<double> pc, Vector3<double> pv, double angle, const Triangle *scene_Triangles, int nTriangles, LightParams *lights, int lights_num) {
    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;

    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;

    double dw = 2.0 / (w - 1.0);
    double dh = 2.0 / (h - 1.0);
    double z = 1.0 / tan(angle * M_PI / 360.0);
    Vector3<double> bz = normalize(diff(pv, pc));
    Vector3<double> bx = normalize(cross_product(bz, {0.0, 0.0, 1.0}));
    Vector3<double> by = normalize(cross_product(bx, bz));
    for (int j = id_y; j < h; j += offset_y)
        for (int i = id_x; i < w; i += offset_x) {
            Vector3<double> v = {-1.0 + dw * i, (-1.0 + dh * j) * h / w, z};
            Vector3<double> dir = mult(bx, by, bz, v);
            data[(h - 1 - j) * w + i] =
                ray(pc, normalize(dir), scene_Triangles, nTriangles, lights, lights_num);
        }
}

__host__ __device__ uchar4 SSAA(uchar4 *data, int i, int j, int w, int h, int kernel_w, int kernel_h) {
    Vector3<double> res;
    for (int y = i; y < i + kernel_h; ++y)
        for (int x = j; x < j + kernel_w; ++x) {
            auto pix = data[y * w + x];
            res = add(res, Vector3<double>{(double)pix.x, (double)pix.y, (double)pix.z});
        }
    auto pix = div(res, kernel_w * kernel_h);
    return make_uchar4(pix.x, pix.y, pix.z, 0);
}

__global__ void getSSAA_CUDA(uchar4 *dst, uchar4 *src, int new_w, int new_h, int w, int h) {
    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;

    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;

    int kernel_w = w / new_w;
    int kernel_h = h / new_h;

    for (int i = id_y; i < new_h; i += offset_y) {
        for (int j = id_x; j < new_w; j += offset_x) {
            int pix_i = i * kernel_h;
            int pix_j = j * kernel_w;

            dst[i * new_w + j] = SSAA(src, pix_i, pix_j, w, h, kernel_w, kernel_h);
        }
    }
}

void getSSAA_OMP(uchar4 *dst, uchar4 *src, int new_w, int new_h, int w, int h) {
    int kernel_w = w / new_w;
    int kernel_h = h / new_h;
#pragma omp parallel for
    for (int pix = 0; pix < new_w * new_h; ++pix) {
        int i = pix / new_w;
        int j = pix % new_w;

        int pix_i = i * kernel_h;
        int pix_j = j * kernel_w;

        dst[i * new_w + j] = SSAA(src, pix_i, pix_j, w, h, kernel_w, kernel_h);
    }
}

void getCameraPos(const CameraMovement &c, const CameraMovement &n, double t, Vector3<double> *pc, Vector3<double> *pv) {
    double phic = c.phi0 + c.wphi * t, phin = n.phi0 + n.wphi * t;
    double rc = c.r0 + c.ar * sin(c.wr * t + c.pr), zc = c.z0 + c.ar * sin(c.wz * t + c.pz);
    double rn = n.r0 + n.ar * sin(n.wr * t + n.pr), zn = n.z0 + n.ar * sin(n.wz * t + n.pz);

    *pv = Vector3<double>{rn * cos(phin), rn * sin(phin), zn};
    *pc = Vector3<double>{rc * cos(phic), rc * sin(phic), zc};
}

void write_image(const std::string &path, const std::vector<uchar4> &data, int w, int h) {
    MPI_File file;
    CHECK_MPI(MPI_File_open(MPI_COMM_SELF, path.data(), MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &file));
    CHECK_MPI(MPI_File_write(file, &w, 1, MPI_INT, MPI_STATUS_IGNORE));
    CHECK_MPI(MPI_File_write(file, &h, 1, MPI_INT, MPI_STATUS_IGNORE));
    CHECK_MPI(MPI_File_write(file, data.data(), sizeof(uchar4) * w * h,  MPI_BYTE, MPI_STATUS_IGNORE));
    CHECK_MPI(MPI_File_close(&file));
}

void signal_handler(int signal){
    std::cout << "Error. Bad signal: " << signal << std::endl;
    MPI_Finalize();
    exit(0);
}

int main(int argc, char *argv[]) {

    std::signal(SIGSEGV, signal_handler);
    std::signal(SIGABRT, signal_handler);

    MPIContext ctx(&argc, &argv);
    parse_flags(argc, argv);

    int rank, nprocesses;
    CHECK_MPI(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    CHECK_MPI(MPI_Comm_size(MPI_COMM_WORLD, &nprocesses));

    CHECK_MPI(MPI_Barrier(MPI_COMM_WORLD));
    std::vector<Triangle> scene_Triangles;
    Params params;
    if (rank == 0) {
        std::cin >> params;
        importObj(scene_Triangles, "hex.obj", params.hex);
        importObj(scene_Triangles, "octa.obj", params.octa);
        importObj(scene_Triangles, "icos.obj", params.icos);
        scene_Triangles.push_back({params.floor.c, params.floor.b, params.floor.a, params.floor.color});
        scene_Triangles.push_back({params.floor.a, params.floor.d, params.floor.c, params.floor.color});
    }

    int output_pattern_size = params.output_pattern.size();
    CHECK_MPI(MPI_Bcast(&params.nframes, 1, MPI_INT, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&output_pattern_size, 1, MPI_INT, 0, MPI_COMM_WORLD));
    params.output_pattern.resize(output_pattern_size);
    CHECK_MPI(MPI_Bcast((char *)params.output_pattern.data(), output_pattern_size, MPI_CHAR, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.w, 1, MPI_INT, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.h, 1, MPI_INT, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.angle, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.camera_center, sizeof(CameraMovement), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.camera_dir, sizeof(CameraMovement), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.hex, sizeof(FigureParams), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.octa, sizeof(FigureParams), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.icos, sizeof(FigureParams), MPI_BYTE, 0, MPI_COMM_WORLD));

    CHECK_MPI(MPI_Bcast(&params.floor.a, sizeof(Vector3<double>), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.floor.b, sizeof(Vector3<double>), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.floor.c, sizeof(Vector3<double>), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.floor.d, sizeof(Vector3<double>), MPI_BYTE, 0, MPI_COMM_WORLD));
    int texture_path_size = params.floor.texture_path.size();
    CHECK_MPI(MPI_Bcast(&texture_path_size, 1, MPI_INT, 0, MPI_COMM_WORLD));
    params.floor.texture_path.resize(texture_path_size);
    CHECK_MPI(MPI_Bcast((char *)params.floor.texture_path.data(), texture_path_size, MPI_CHAR, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.floor.color, sizeof(Vector3<double>), MPI_BYTE, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(&params.floor.k_refl, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD));

    params.lights.resize(params.lights_num);
    CHECK_MPI(MPI_Bcast(&params.lights_num, 1, MPI_INT, 0, MPI_COMM_WORLD));
    CHECK_MPI(MPI_Bcast(params.lights.data(), sizeof(LightParams) * params.lights_num, MPI_BYTE, 0, MPI_COMM_WORLD));

    int nTriangles = scene_Triangles.size();
    CHECK_MPI(MPI_Bcast(&nTriangles, 1, MPI_INT, 0, MPI_COMM_WORLD));
    scene_Triangles.resize(nTriangles);
    CHECK_MPI(MPI_Bcast(scene_Triangles.data(), sizeof(Triangle) * nTriangles, MPI_BYTE, 0, MPI_COMM_WORLD));

    int ndevices;
    CSC(hipGetDeviceCount(&ndevices));
    CSC(hipSetDevice(rank % ndevices));

    Triangle *gpu_scene_Triangles;
    LightParams *gpu_lights;
    if (parallelizationMode == CUDA) {
        auto Triangles_size = sizeof(Triangle) * scene_Triangles.size();
        CSC(hipMalloc(&gpu_scene_Triangles, Triangles_size));
        CSC(hipMemcpy(gpu_scene_Triangles, scene_Triangles.data(), Triangles_size, hipMemcpyHostToDevice));
        auto lights_size = sizeof(LightParams) * params.lights_num;
        CSC(hipMalloc(&gpu_lights, lights_size));
        CSC(hipMemcpy(gpu_lights, params.lights.data(), lights_size, hipMemcpyHostToDevice));
    }

    CHECK_MPI(MPI_Barrier(MPI_COMM_WORLD));
    std::vector<uchar4> data_render(SSSAx*params.w * SSSAx*params.h), data_ssaa(params.w * params.h);
    uchar4 *gpu_data_render, *gpu_data_ssaa;
    if (parallelizationMode == CUDA) {
        CSC(hipMalloc(&gpu_data_render, sizeof(uchar4) * SSSAx*params.w * SSSAx*params.h));
        CSC(hipMalloc(&gpu_data_ssaa, sizeof(uchar4) * params.w * params.h));
    }
    for (int frame = rank; frame < params.nframes; frame += nprocesses) {
        Vector3<double> pc, pv;
        getCameraPos(params.camera_center, params.camera_dir, 0.1 * (double)frame, &pc, &pv);
        auto start = std::chrono::high_resolution_clock::now();

        if (parallelizationMode == OpenMP) {
            getRenderOMP(data_render.data(), SSSAx*params.w, SSSAx*params.h, pc, pv, params.angle, scene_Triangles.data(), scene_Triangles.size(), params.lights.data(), params.lights.size());
            getSSAA_OMP(data_ssaa.data(), data_render.data(), params.w, params.h, SSSAx*params.w, SSSAx*params.h);
        }

        if (parallelizationMode == CUDA) {
            getRenderCUDA<<<NBLOCKS, NTHREADS>>>(gpu_data_render, SSSAx*params.w, SSSAx*params.h, pc, pv, params.angle,gpu_scene_Triangles, scene_Triangles.size(), gpu_lights, params.lights.size());
            CSC(hipDeviceSynchronize());
            getSSAA_CUDA<<<NBLOCKS, NTHREADS>>>(gpu_data_ssaa, gpu_data_render, params.w, params.h, SSSAx*params.w, SSSAx*params.h);
            CSC(hipMemcpy(data_ssaa.data(), gpu_data_ssaa,sizeof(uchar4) * params.w * params.h, hipMemcpyDeviceToHost));
        }

        char output_path[256];
        sprintf(output_path, params.output_pattern.data(), frame);
        write_image(output_path, data_ssaa, params.w, params.h);
        auto time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start);
        std::cout << frame << "\t" << output_path << "\t" << time.count() << "ms" << std::endl;
    }

    if (parallelizationMode == CUDA) {
        CSC(hipFree(gpu_scene_Triangles));
        CSC(hipFree(gpu_lights));
        CSC(hipFree(gpu_data_ssaa));
        CSC(hipFree(gpu_data_render));
    }

    return 0;
}